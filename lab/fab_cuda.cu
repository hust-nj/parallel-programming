
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

long long res[64];

__global__ void fib(long long *res)
{
    int idx = threadIdx.x;
    res[idx] = (long long)(1.0/sqrt(5.0)*(pow((1+sqrt(5.0))/2.0, idx+1) - pow((1-sqrt(5.0))/2.0, idx+1)) + 0.5);
    // printf("%d\n", res[idx]);
}

int main()
{
    int n;
    long long *gpures;
    scanf("%d", &n);
    hipMalloc(&gpures, n*sizeof(long long));
    fib<<<1,n>>>(gpures);
    hipMemcpy(res, gpures, n*sizeof(long long), hipMemcpyDeviceToHost);
    hipFree(gpures);
    for (int i = 0; i < n; ++i)
    {
        printf(i == n-1 ? "%lld\n" : "%lld ", res[i]);
    }
    return 0;
}